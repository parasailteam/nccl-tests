/*************************************************************************
 * Copyright (c) 2016-2019, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

//make VERBOSE=10 NCCL_HOME=`pwd`/../build MPI=1 MPI_HOME=/usr/local/include/openmpi/ -j

#include "hip/hip_runtime.h"
#include "common.h"

void print_header() {
  PRINT("# %10s  %12s  %6s  %6s            out-of-place                       in-place          \n", "", "", "", "");
  PRINT("# %10s  %12s  %6s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "size", "count", "type", "redop",
        "time", "algbw", "busbw", "error", "time", "algbw", "busbw", "error");
  PRINT("# %10s  %12s  %6s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "", "",
        "(us)", "(GB/s)", "(GB/s)", "", "(us)", "(GB/s)", "(GB/s)", "");
}

void print_line_header (size_t size, size_t count, const char *typeName, const char *opName, int root) {
  PRINT("%12li  %12li  %6s  %6s", size, count, typeName, opName);
}

void CustomCollectiveGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count;
  *recvcount = count;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = *sendcount;
}

testResult_t CustomCollectiveInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;
  /**For now this custom collective only supports checking results for AllReduce*/
  for (int i=0; i<args->nGpus; i++) {
    int gpuid = args->localRank*args->nThreads*args->nGpus + args->thread*args->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    TESTCHECK(InitData(args->recvbuffs[i], sendcount, type, rep, rank));
    TESTCHECK(InitData(args->sendbuffs[i], sendcount, type, rep, rank));
    if (rank == 0)
      TESTCHECK(InitData(args->expected[i], sendcount, type, rep, 0));
    else
      TESTCHECK(InitData(args->expected[i], sendcount, type, rep, rank-1));
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void CustomCollectiveGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(2*(nranks - 1)))/((double)nranks);
  *busBw = baseBw * factor;
}

testResult_t CustomCollectiveRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  int rank, nranks;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  NCCLCHECK(ncclCommCount(comm, &nranks));
  NCCLCHECK(ncclGroupStart());
  if (rank < nranks-1)
    NCCLCHECK(ncclSend(((char*)sendbuff), count, type, rank+1, comm, stream));
  if (rank > 0)
    NCCLCHECK(ncclRecv(((char*)recvbuff), count, type, rank-1, comm, stream));
  NCCLCHECK(ncclGroupEnd());
 // NCCLCHECK(ncclCustomCollective(sendbuff, recvbuff, count, type, 0, comm, stream));
  return testSuccess;
}

struct testColl customCollTest = {
  "CustomCollective",
  CustomCollectiveGetCollByteCount,
  CustomCollectiveInitData,
  CustomCollectiveGetBw,
  CustomCollectiveRunColl
};

void CustomCollectiveGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  CustomCollectiveGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t CustomCollectiveRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &customCollTest;
  ncclDataType_t *run_types;
  ncclRedOp_t *run_ops;
  const char **run_typenames, **run_opnames;
  int type_count, op_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = ncclNumTypes;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if ((int)op != -1) {
    op_count = 1;
    run_ops = &op;
    run_opnames = &opName;
  } else {
    op_count = ncclNumOps;
    run_ops = test_ops;
    run_opnames = test_opnames;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=0; j<op_count; j++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], run_ops[j], run_opnames[j], -1));
    }
  }
  return testSuccess;
}

struct testEngine CustomCollectiveEngine = {
  CustomCollectiveGetBuffSize,
  CustomCollectiveRunTest,
};

#pragma weak ncclTestEngine=CustomCollectiveEngine
